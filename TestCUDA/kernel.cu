#include "hip/hip_runtime.h"
#include "pch.h"
#include "CudaMemory.h"
#include "CudaManagedMemory.h"
#include "Timer.h"
#include "CudaDevice.h"

__global__ void add(int count, float* x, float* y)
{
	for (auto index = blockIdx.x * blockDim.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x)
		y[index] = x[index] + y[index];
}

void TestDevice()
{
	const auto count = 1 << 20;
	const auto size = static_cast<int>(count * sizeof(float));

	auto x = make_unique<float[]>(count);
	auto y = make_unique<float[]>(count);

	fill(x.get(), x.get() + count, 1.0f);
	fill(y.get(), y.get() + count, 2.0f);

	CudaMemory xDevice{ size, x.get() };
	CudaMemory yDevice{ size, y.get() };

	Timer miniTimer;

	auto maxThreadsPerBlock = 1024;
	auto numBlocks = (count + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
	add<<<numBlocks, maxThreadsPerBlock>>>(count, xDevice.Get<float*>(), yDevice.Get<float*>());
	CudaDevice::Synchronize();

	yDevice.CopyToHost(y.get(), size);

	cout << "Mini-duration: " << miniTimer.GetDurationMs() << "ms\n";

	float maxError = 0.0f;
	for (auto index = 0; index < count; ++index)
		maxError = max(maxError, fabs(y[index] - 3.0f));
	cout << "Max error: " << maxError << '\n';
}

void TestManaged()
{
	const auto count = 1 << 20;
	const auto size = static_cast<int>(count * sizeof(float));

	CudaManagedMemory xDevice{ size };
	CudaManagedMemory yDevice{ size };
	auto x = xDevice.Get<float*>();
	auto y = yDevice.Get<float*>();

	fill(x, x + count, 1.0f);
	fill(y, y + count, 2.0f);

	Timer miniTimer;

	auto maxThreadsPerBlock = 1024;
	auto numBlocks = (count + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
	add<<<numBlocks, maxThreadsPerBlock>>>(count, x, y);
	CudaDevice::Synchronize();

	cout << "Mini-duration: " << miniTimer.GetDurationMs() << "ms\n";

	float maxError = 0.0f;
	for (auto index = 0; index < count; ++index)
		maxError = max(maxError, fabs(y[index] - 3.0f));
	cout << "Max error: " << maxError << '\n';
}



#if 0
vector<pair<int, int>> CreateTwoCards()
{
	return
	{
		{ 2, 3 }, { 2, 4 }, { 2, 5 }, { 2, 7 }, { 2, 8 }, { 2, 9 }, { 2, 10 }, { 2, 11 }, { 2, 12 },
	{ 3, 4 }, { 3, 5 }, { 3, 7 }, { 3, 8 }, { 3, 9 }, { 3, 10 }, { 3, 11 }, { 3, 12 },
	{ 4, 5 }, { 4, 7 }, { 4, 8 }, { 4, 9 }, { 4, 10 }, { 4, 11 }, { 4, 12 },
	{ 5, 7 }, { 5, 8 }, { 5, 9 }, { 5, 10 }, { 5, 11 }, { 5, 12 }, { 5, 33 }, { 5, 46 },
	{ 7, 8 }, { 7, 9 }, { 7, 10 }, { 7, 11 }, { 7, 12 }, { 7, 18 }, { 7, 23 }, { 7, 31 }, { 7, 36 }, { 7, 44 }, { 7, 49 },
	{ 8, 9 }, { 8, 10 }, { 8, 11 }, { 8, 12 }, { 8, 13 }, { 8, 14 }, { 8, 19 }, { 8, 21 },
	{ 8, 22 }, { 8, 26 }, { 8, 27 }, { 8, 32 }, { 8, 39 }, { 8, 40 }, { 8, 45 }, { 8, 47 },
	{ 9, 10 }, { 9, 11 }, { 9, 12 }, { 9, 13 }, { 9, 14 }, { 9, 19 }, { 9, 21 }, { 9, 22 },
	{ 9, 23 }, { 9, 24 }, { 9, 25 }, { 9, 26 }, { 9, 27 }, { 9, 32 }, { 9, 36 }, { 9, 37 },
	{ 9, 38 }, { 9, 39 }, { 9, 40 }, { 9, 45 }, { 9, 47 }, { 9, 49 }, { 9, 50 }, { 9, 51 },
	{ 10, 11 }, { 10, 12 }, { 10, 22 }, { 10, 23 }, { 10, 33 }, { 10, 36 }, { 10, 46 }, { 10, 49 },
	{ 11, 12 }, { 11, 22 }, { 11, 24 }, { 11, 37 }, { 11, 50 },
	{ 12, 22 }, { 12, 25 }, { 12, 38 }, { 12, 51 },
	{ 13, 14 }, { 13, 19 }, { 13, 21 }, { 13, 22 }, { 13, 26 }, { 13, 27 }, { 13, 32 }, { 13, 39 }, { 13, 40 }, { 13, 45 }, { 13, 47 },
	{ 14, 19 }, { 14, 21 }, { 14, 22 }, { 14, 26 }, { 14, 27 }, { 14, 32 }, { 14, 39 }, { 14, 40 }, { 14, 45 }, { 14, 47 },
	{ 18, 33 }, { 18, 46 },
	{ 19, 21 }, { 19, 22 }, { 19, 26 }, { 19, 27 }, { 19, 32 }, { 19, 39 }, { 19, 40 }, { 19, 45 }, { 19, 47 },
	{ 21, 22 }, { 21, 26 }, { 21, 27 }, { 21, 32 }, { 21, 39 }, { 21, 40 }, { 21, 45 }, { 21, 47 },
	{ 22, 23 }, { 22, 24 }, { 22, 25 }, { 22, 26 }, { 22, 27 }, { 22, 32 }, { 22, 36 }, { 22, 37 },
	{ 22, 38 }, { 22, 39 }, { 22, 40 }, { 22, 45 }, { 22, 47 }, { 22, 49 }, { 22, 50 }, { 22, 51 },
	{ 23, 33 }, { 23, 36 }, { 23, 46 }, { 23, 49 },
	{ 24, 37 }, { 24, 50 },
	{ 25, 38 }, { 25, 51 },
	{ 26, 27 }, { 26, 32 }, { 26, 39 }, { 26, 40 }, { 26, 45 }, { 26, 47 },
	{ 27, 32 }, { 27, 39 }, { 27, 40 }, { 27, 45 }, { 27, 47 },
	{ 31, 33 }, { 31, 46 },
	{ 32, 39 }, { 32, 40 }, { 32, 45 }, { 32, 47 },
	{ 33, 36 }, { 33, 44 }, { 33, 49 },
	{ 36, 46 }, { 36, 49 },
	{ 37, 50 },
	{ 38, 51 },
	{ 39, 40 }, { 39, 45 }, { 39, 47 },
	{ 40, 45 }, { 40, 47 },
	{ 44, 46 },
	{ 45, 47 },
	{ 46, 49 }
	};
}

LargeInteger Combinations(LargeInteger total, LargeInteger take)
{
	LargeInteger result{ 1 };
	for (LargeInteger index{ 0 }; index < take; ++index)
		result = result * (total - index) / (index + 1);
	return result;
}

LargeInteger Partitions(LargeInteger count)
{
	auto n = count / 2;
	LargeInteger product{ 1 };
	for (LargeInteger k{ 0 }; k < n; ++k)
		product *= 2 * k + 1;
	return product;
}

LargeInteger ComputeTotalCombinations(LargeInteger cards, LargeInteger opponentCards)
{
	if (opponentCards < 0)
		return 0;
	if (opponentCards == 0)
		return 1;
	return Combinations(cards, opponentCards) * Partitions(opponentCards);
}

class TwoCardOverlap
{
public:
	TwoCardOverlap() = default;
	TwoCardOverlap(const pair<int, int>& hole)
		: bits{ (1ull << hole.first) | (1ull << hole.second) }
	{
	}
	TwoCardOverlap(const TwoCardOverlap& rhs) = default;
	TwoCardOverlap(TwoCardOverlap&& rhs) = default;
	~TwoCardOverlap() = default;

	TwoCardOverlap& operator=(const TwoCardOverlap& rhs) = default;
	TwoCardOverlap& operator=(TwoCardOverlap&& rhs) = default;

	unsigned long long GetBits() const
	{
		return bits;
	}

	void AddCount2()
	{
		++count2;
	}
	void AddCount3()
	{
		++count3;
	}
	void AddCount4()
	{
		++count4;
	}
	void AddCount5()
	{
		++count5;
	}

	LargeInteger GetTotalOverlap(LargeInteger remainingCards, LargeInteger opponentCards) const
	{
		return count2 * ComputeTotalCombinations(remainingCards, opponentCards) -
			count3 * ComputeTotalCombinations(remainingCards - 2, opponentCards - 2) +
			count4 * ComputeTotalCombinations(remainingCards - 4, opponentCards - 4) -
			count5 * ComputeTotalCombinations(remainingCards - 6, opponentCards - 6);
	}

	TwoCardOverlap& operator+=(const TwoCardOverlap& rhs)
	{
		count2 += rhs.count2;
		count3 += rhs.count3;
		count4 += rhs.count4;
		count5 += rhs.count5;
		return *this;
	}

private:
	unsigned long long bits = 0;
	LargeInteger count2 = 0;
	LargeInteger count3 = 0;
	LargeInteger count4 = 0;
	LargeInteger count5 = 0;
};

void Compute5(TwoCardOverlap* begin, TwoCardOverlap* end)
{
	for_each(execution::par_unseq, begin, end, [&](TwoCardOverlap& i1)
	{
		for (auto i2 = begin; i2 != &i1; ++i2)
		{
			if ((i1.GetBits() & i2->GetBits()) != 0)
				continue;
			auto s2 = i1.GetBits() | i2->GetBits();
			i1.AddCount2();
			for (auto i3 = begin; i3 != i2; ++i3)
			{
				if ((s2 & i3->GetBits()) != 0)
					continue;
				auto s3 = s2 | i3->GetBits();
				i1.AddCount3();
				for (auto i4 = begin; i4 != i3; ++i4)
				{
					if ((s3 & i4->GetBits()) != 0)
						continue;
					auto s4 = s3 | i4->GetBits();
					i1.AddCount4();
					for (auto i5 = begin; i5 != i4; ++i5)
						if ((s4 & i5->GetBits()) == 0)
							i1.AddCount5();
				}
			}
		}
	});
}

LargeInteger Compute(const vector<pair<int, int>>& twoCards, int remaining)
{
	vector<TwoCardOverlap> overlaps;
	for (auto& twoCard : twoCards)
		overlaps.emplace_back(twoCard);
	auto begin = overlaps.data();
	auto end = begin + overlaps.size();

	Compute5(begin, end);

	TwoCardOverlap total;
	for (auto& twoCard : overlaps)
		total += twoCard;
	return total.GetTotalOverlap(remaining - 4, 2 * 5 - 4);
}
#endif

//This won't compile with nvcc.
//#include <boost/multiprecision/cpp_int.hpp>
//using LargeInteger = boost::multiprecision::int128_t;

void TestOddsDevice()
{
}
